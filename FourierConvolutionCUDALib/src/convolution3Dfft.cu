#include "hip/hip_runtime.h"
#include "convolution3Dfft.h"
#include "book.h"
#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"

#include <iostream>
#include <cmath>
#include <algorithm>
#include <vector>
#include <numeric>

#include "traits.hpp"
namespace fc = fourierconvolution;

//__device__ static const float PI_2 = 6.28318530717958620f;
//__device__ static const float PI_1 =  3.14159265358979310f;

////////////////////////////////////////////////////////////////////////////////
// Modulate Fourier image of padded data by Fourier image of padded kernel
// and normalize by FFT size
////////////////////////////////////////////////////////////////////////////////
//Adapted from CUDA SDK examples



int closest_multiplier(int _a_number, int _multiplier = 32){

  int value = 1;
  int max = (_a_number + _multiplier -1)/_multiplier;
  int diff = std::abs(~0);
  for(int i = 1;i<=max;++i){
    int temp_diff = _a_number - (i*_multiplier);
    if(std::abs(temp_diff) < diff)
      value = i;
  }

  return value*_multiplier;
}


__device__ void mulAndScale(hipfftComplex& a, const hipfftComplex& b, const float& c)
{
    hipfftComplex t = {c * (a.x * b.x - a.y * b.y), c * (a.y * b.x + a.x * b.y)};
    a = t;
};

__global__ void __launch_bounds__(MAX_THREADS_CUDA)  modulateAndNormalize_kernel(hipfftComplex *d_Dst, hipfftComplex *d_Src, long long int dataSize,float c)
{
    long long int i = (long long int)blockDim.x * (long long int)blockIdx.x + (long long int)threadIdx.x;
	long long int offset = (long long int)blockDim.x * (long long int)gridDim.x;
    while(i < dataSize)
	{		

		hipfftComplex a = d_Src[i];
		hipfftComplex b = d_Dst[i];

		mulAndScale(a, b, c);
		d_Dst[i] = a;

		i += offset;
	}
};

//we use nearest neighbor interpolation to access FFT coefficients in the kernel
__global__ void __launch_bounds__(MAX_THREADS_CUDA)  modulateAndNormalizeSubsampled_kernel(hipfftComplex *d_Dst, hipfftComplex *d_Src,int kernelDim_0,int kernelDim_1,int kernelDim_2,int imDim_0,int imDim_1,int imDim_2,long long int datasize,float c)
{

	float r_0 = ((float)kernelDim_0) / ((float)imDim_0); //ratio between image size and kernel size to calculate access
	float r_1 = ((float)kernelDim_1) / ((float)imDim_1);
	float r_2 = ((float)kernelDim_2) / ((float)imDim_2);

    long long int i = (long long int)blockDim.x * (long long int)blockIdx.x + (long long int)threadIdx.x;
	long long int offset = (long long int)blockDim.x * (long long int)gridDim.x;
	int k_0,k_1,k_2;
	int aux;
	//	float auxExp, auxSin,auxCos;
    while(i < datasize)
	{
		//for each dimension we need to access k_i*r_i  i=0, 1, 2
		aux = 1 + imDim_2/2;
		k_2 = i % aux;
		aux = (i - k_2) / aux;
		k_1 = aux % imDim_1;
		k_0 = (aux - k_1) / imDim_1;

		hipfftComplex b = d_Dst[i];

		//apply shift in fourier domain since we did not apply fftshift to kernel (so we could use the trick of assuming the kernel is padded with zeros and then just subsample FFT)
		/* This is how we would do it in Matlab (linear phase change)
		auxExp = k_0 * r_0;
		auxExp += k_1 * r_1;
		auxExp += k_2 * r_2;
		auxExp *= PI_1;
		auxSin = sin(auxExp);
		auxCos = cos(auxExp);
		auxExp = b.x * auxCos - b.y * auxSin;

		b.y = b.x * auxSin + b.y * auxCos;
		b.x = auxExp;
		*/

		//add the ratio to each dimension and apply nearest neighbor interpolation
		//k_2 = min((int)(r_2*(float)k_2 + 0.5f),kernelDim_2-1);//the very end points need to be interpolated as "ceiling" instead of round or we can get oout of bounds access
		//k_1 = min((int)(r_1*(float)k_1 + 0.5f),kernelDim_1-1);
		//k_0 = min((int)(r_0*(float)k_0 + 0.5f),kernelDim_0-1);
		k_2 = ((int)(r_2*(float)k_2 + 0.5f)) % kernelDim_2;//the very end points need to be interpolated as "ceiling" instead of round or we can get oout of bounds access
		k_1 = ((int)(r_1*(float)k_1 + 0.5f)) % kernelDim_1;
		k_0 = ((int)(r_0*(float)k_0 + 0.5f)) % kernelDim_0;
		//calculate new coordinate relative to kernel size
		aux = 1 + kernelDim_2/2;
		hipfftComplex a = d_Src[k_2 + aux *(k_1 + kernelDim_1 * k_0)];
		
		if( (k_0 + k_1 + k_2) % 2 == 1 )//after much debugging it seems the phase shift is 0 or Pi (nothing in between). In Matlab is a nice linear change as programmed above
		{
			a.x = -a.x;
			a.y = -a.y;
		}
		mulAndScale(a, b, c);

		//__syncthreads();//this actually slows down the code by a lot (0.1 sec for 512x512x512)
		d_Dst[i] = a;

		i += offset;
	}
};

//WARNING: for cuFFT the fastest running index is z direction!!! so pos = z + imDim[2] * (y + imDim[1] * x)
__global__ void __launch_bounds__(MAX_THREADS_CUDA) fftShiftKernel(imageType* kernelCUDA,
								   imageType* kernelPaddedCUDA,
								   int kernelDim_0,
								   int kernelDim_1,
								   int kernelDim_2,
								   int imDim_0,
								   int imDim_1,
								   int imDim_2)
{
	int kernelSize = kernelDim_0 * kernelDim_1 * kernelDim_2;

	int tid = blockDim.x * blockIdx.x + threadIdx.x;

	if(tid<kernelSize)
	{
		//find coordinates
		long long int x,y,z,aux;
		z = tid % kernelDim_2;
		aux = (tid - z)/kernelDim_2;
		y = aux % kernelDim_1;
		x = (aux - y)/kernelDim_1;

		//center coordinates
		x -= kernelDim_0/2;
		y -= kernelDim_1/2;
		z -= kernelDim_2/2;

		//circular shift if necessary
		if(x<0) x += imDim_0;
		if(y<0) y += imDim_1;
		if(z<0) z += imDim_2;

		//calculate position in padded kernel
		aux = z + imDim_2 * (y + imDim_1 * x);

		//copy value
		kernelPaddedCUDA[aux] = kernelCUDA[tid];//for the most part it should be a coalescent access in oth places
	}
}

//=====================================================================
//-------------to debug elements--------------------------------------
void writeOutCUDAfft(char* filename,imageType* fftCUDA,int* fftCUDAdims)
{
	int fftSize = 1;
	for(int ii=0;ii<dimsImage;ii++)
	{
		fftSize *= fftCUDAdims[ii];
	}

	//copy FFT from CUDA
	imageType* fftHOST = new imageType[2*fftSize];//complex format
	HANDLE_ERROR(hipMemcpy(fftHOST,fftCUDA,2*sizeof(imageType)*fftSize,hipMemcpyDeviceToHost));

	//calculate module
	/*
	int count = 0;
	for(int ii=0;ii<fftSize;ii++)
	{
		fftHOST[ii] = sqrt(fftHOST[count]*fftHOST[count] + fftHOST[count+1]*fftHOST[count+1]);
		count += 2;
	}
	*/

	FILE* fid = fopen(filename,"wb");
	if(fid == NULL)
	{
		printf("ERROR: at writeOutCUDAfft opening file %s\n",filename);
		exit(2);
	}else{
		printf("DEBUGGING: Writing FFT (real part first,imaginary second)  from CUDA of dimensions %d x %d x %d in file %s\n",fftCUDAdims[2],fftCUDAdims[1],fftCUDAdims[0],filename);
	}
	//fwrite(fftHOST,sizeof(imageType),fftSize,fid);
	for(int ii=0;ii<2*fftSize;ii+=2)
		fwrite(&(fftHOST[ii]),sizeof(imageType),1,fid);
	for(int ii=1;ii<2*fftSize;ii+=2)
		fwrite(&(fftHOST[ii]),sizeof(imageType),1,fid);


	fclose(fid);
	delete[] fftHOST;
}


//=====================================================================
//WARNING: for cuFFT the fastest running index is z direction!!! so pos = z + imDim[2] * (y + imDim[1] * x)

//WARNING: the following function assumes the old cufft native format for the array and has not yet been ported to fftw style layout
imageType* convolution3DfftCUDA_test(imageType* im,
				     int* imDim,
				     imageType* kernel,
				     int devCUDA)
{
	imageType* convResult = NULL;
	imageType* imCUDA = NULL;
	imageType* kernelCUDA = NULL;


	hipfftHandle fftPlanFwd, fftPlanInv;

	
	HANDLE_ERROR( hipSetDevice( devCUDA ) );

	long long int imSize = 1;
	for(int ii=0;ii<dimsImage;ii++)
	{
		imSize *= (long long int) (imDim[ii]);
	}

	long long int imSizeFFT = imSize+(long long int)(2*imDim[0]*imDim[1]); //size of the R2C transform in cuFFTComplex

	//allocate memory for output result
	convResult = new imageType[imSize];

	//allocat ememory in GPU
	HANDLE_ERROR( hipMalloc( (void**)&(imCUDA), imSizeFFT*sizeof(imageType) ) );//a little bit larger to allow in-place FFT
	HANDLE_ERROR( hipMalloc( (void**)&(kernelCUDA), imSizeFFT*sizeof(imageType) ) );


	//TODO: pad image to a power of 2 size in all dimensions (use whatever  boundary conditions you want to apply)
	//TODO: pad kernel to image size
	//TODO: pad kernel and image to xy(z/2 + 1) for in-place transform
	//NOTE: in the example for 2D convolution using FFT in the Nvidia SDK they do the padding in the GPU, but in might be pushing the memory in the GPU for large images.

	//printf("Copying memory (kernel and image) to GPU\n");
	HANDLE_ERROR( hipMemcpy( kernelCUDA, kernel, imSize*sizeof(imageType) , hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy( imCUDA, im, imSize*sizeof(imageType) , hipMemcpyHostToDevice ) );
	
	//printf("Creating R2C & C2R FFT plans for size %i x %i x %i\n",imDim[0],imDim[1],imDim[2]);
	CUFFT_ERROR(hipfftPlan3d(&fftPlanFwd, imDim[0], imDim[1], imDim[2], HIPFFT_R2C));
	CUFFT_ERROR(hipfftPlan3d(&fftPlanInv, imDim[0], imDim[1], imDim[2], HIPFFT_C2R));

#if CUDART_VERSION <= 7500
	CUFFT_ERROR(cufftSetCompatibilityMode(fftPlanFwd,CUFFT_COMPATIBILITY_NATIVE)); //for highest performance since we do not need FFTW compatibility
	CUFFT_ERROR(cufftSetCompatibilityMode(fftPlanInv,CUFFT_COMPATIBILITY_NATIVE));
#endif

	//transforming convolution kernel; TODO: if I do multiple convolutions with the same kernel I could reuse the results at teh expense of using out-of place memory (and then teh layout of the data is different!!!! so imCUDAfft should also be out of place)
	//NOTE: from CUFFT manual: If idata and odata are the same, this method does an in-place transform.
	//NOTE: from CUFFT manual: inplace output data xy(z/2 + 1) with fcomplex. Therefore, in order to perform an in-place FFT, the user has to pad the input array in the last dimension to Nn2 + 1 complex elements interleaved. Note that the real-to-complex transform is implicitly forward.
	hipfftExecR2C(fftPlanFwd, imCUDA, (hipfftComplex *)imCUDA);HANDLE_ERROR_KERNEL;
	//transforming image
	hipfftExecR2C(fftPlanFwd, kernelCUDA, (hipfftComplex *)kernelCUDA);HANDLE_ERROR_KERNEL;
	

	//multiply image and kernel in fourier space (and normalize)
	//NOTE: from CUFFT manual: CUFFT performs un-normalized FFTs; that is, performing a forward FFT on an input data set followed by an inverse FFT on the resulting set yields data that is equal to the input scaled by the number of elements.
	int numThreads=std::min((long long int)MAX_THREADS_CUDA,imSizeFFT/2);//we are using complex number
	int numBlocks=std::min((long long int)MAX_BLOCKS_CUDA,(long long int)(imSizeFFT/2+(long long int)(numThreads-1))/((long long int)numThreads));
	modulateAndNormalize_kernel<<<numBlocks,numThreads>>>((hipfftComplex *)imCUDA, (hipfftComplex *)kernelCUDA, imSizeFFT/2,1.0f/(float)(imSize));//last parameter is the size of the FFT

	//inverse FFT 
	hipfftExecC2R(fftPlanInv, (hipfftComplex *)imCUDA, imCUDA);HANDLE_ERROR_KERNEL;

	//copy result to host
	HANDLE_ERROR(hipMemcpy(convResult,imCUDA,sizeof(imageType)*imSize,hipMemcpyDeviceToHost));

	//release memory
	( hipfftDestroy(fftPlanInv) );HANDLE_ERROR_KERNEL;
    ( hipfftDestroy(fftPlanFwd) );HANDLE_ERROR_KERNEL;
	HANDLE_ERROR( hipFree( imCUDA));
	HANDLE_ERROR( hipFree( kernelCUDA));

	return convResult;
}

//=====================================================================
//WARNING: for cuFFT the fastest running index is z direction!!! so pos = z + imDim[2] * (y + imDim[1] * x)
//NOTE: to avoid transferring a large padded kernel, since memcpy is a limiting factor

//WARNING: the following function assumes the old cufft native format for the array and has not yet been ported to fftw style layout
 imageType* convolution3DfftCUDA(imageType* im,
				 int* imDim,
				 imageType* kernel,
				 int* kernelDim,
				 int devCUDA)
{
	imageType* convResult = NULL;
	imageType* imCUDA = NULL;
	imageType* kernelCUDA = NULL;
	imageType* kernelPaddedCUDA = NULL;


	hipfftHandle fftPlanFwd, fftPlanInv;

	
	HANDLE_ERROR( hipSetDevice( devCUDA ) );

	long long int imSize = 1;
	long long int kernelSize = 1;
	for(int ii=0;ii<dimsImage;ii++)
	{
		imSize *= (long long int) (imDim[ii]);
		kernelSize *= (long long int) (kernelDim[ii]);
	}

	long long int imSizeFFT = imSize+(long long int)(2*imDim[0]*imDim[1]); //size of the R2C transform in cuFFTComplex

	//allocate memory for output result
	convResult = new imageType[imSize];

	//allocat ememory in GPU
	HANDLE_ERROR( hipMalloc( (void**)&(imCUDA), imSizeFFT*sizeof(imageType) ) );//a little bit larger to allow in-place FFT
	HANDLE_ERROR( hipMalloc( (void**)&(kernelCUDA), (kernelSize)*sizeof(imageType) ) );
	HANDLE_ERROR( hipMalloc( (void**)&(kernelPaddedCUDA), imSizeFFT*sizeof(imageType) ) );


	//TODO: pad image to a power of 2 size in all dimensions (use whatever  boundary conditions you want to apply)
	//TODO: pad kernel to image size
	//TODO: pad kernel and image to xy(z/2 + 1) for in-place transform
	//NOTE: in the example for 2D convolution using FFT in the Nvidia SDK they do the padding in the GPU, but in might be pushing the memory in the GPU for large images.

	//printf("Copying memory (kernel and image) to GPU\n");
	HANDLE_ERROR( hipMemcpy( kernelCUDA, kernel, kernelSize*sizeof(imageType) , hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy( imCUDA, im, imSize*sizeof(imageType) , hipMemcpyHostToDevice ) );

	//apply ffshift to kernel and pad it with zeros so we can calculate convolution with FFT
	HANDLE_ERROR( hipMemset( kernelPaddedCUDA, 0, imSizeFFT*sizeof(imageType) ));
	int numThreads=std::min((long long int)MAX_THREADS_CUDA,kernelSize);
	int numBlocks=std::min((long long int)MAX_BLOCKS_CUDA,(long long int)(kernelSize+(long long int)(numThreads-1))/((long long int)numThreads));
	fftShiftKernel<<<numBlocks,numThreads>>>(kernelCUDA,kernelPaddedCUDA,kernelDim[0],kernelDim[1],kernelDim[2],imDim[0],imDim[1],imDim[2]);HANDLE_ERROR_KERNEL;

	
	//make sure GPU finishes before we launch two different streams
	HANDLE_ERROR(hipDeviceSynchronize());	

	//printf("Creating R2C & C2R FFT plans for size %i x %i x %i\n",imDim[0],imDim[1],imDim[2]);
	CUFFT_ERROR(hipfftPlan3d(&fftPlanFwd, imDim[0], imDim[1], imDim[2], HIPFFT_R2C));
	CUFFT_ERROR(hipfftPlan3d(&fftPlanInv, imDim[0], imDim[1], imDim[2], HIPFFT_C2R));

#if CUDART_VERSION <= 7500
	CUFFT_ERROR(cufftSetCompatibilityMode(fftPlanFwd,CUFFT_COMPATIBILITY_NATIVE)); //for highest performance since we do not need FFTW compatibility
	CUFFT_ERROR(cufftSetCompatibilityMode(fftPlanInv,CUFFT_COMPATIBILITY_NATIVE));
#endif
	
	//transforming convolution kernel; TODO: if I do multiple convolutions with the same kernel I could reuse the results at teh expense of using out-of place memory (and then teh layout of the data is different!!!! so imCUDAfft should also be out of place)
	//NOTE: from CUFFT manual: If idata and odata are the same, this method does an in-place transform.
	//NOTE: from CUFFT manual: inplace output data xy(z/2 + 1) with fcomplex. Therefore, in order to perform an in-place FFT, the user has to pad the input array in the last dimension to Nn2 + 1 complex elements interleaved. Note that the real-to-complex transform is implicitly forward.
	CUFFT_ERROR(hipfftExecR2C(fftPlanFwd, imCUDA, (hipfftComplex *)imCUDA));
	//transforming image
	CUFFT_ERROR(hipfftExecR2C(fftPlanFwd, kernelPaddedCUDA, (hipfftComplex *)kernelPaddedCUDA));
	

	//multiply image and kernel in fourier space (and normalize)
	//NOTE: from CUFFT manual: CUFFT performs un-normalized FFTs; that is, performing a forward FFT on an input data set followed by an inverse FFT on the resulting set yields data that is equal to the input scaled by the number of elements.
	numThreads=std::min((long long int)MAX_THREADS_CUDA,imSizeFFT/2);//we are using complex numbers
	numBlocks=std::min((long long int)MAX_BLOCKS_CUDA,(long long int)(imSizeFFT/2+(long long int)(numThreads-1))/((long long int)numThreads));
	modulateAndNormalize_kernel<<<numBlocks,numThreads>>>((hipfftComplex *)imCUDA, (hipfftComplex *)kernelPaddedCUDA, imSizeFFT/2,1.0f/(float)(imSize));//last parameter is the size of the FFT

	//inverse FFT 
	hipfftExecC2R(fftPlanInv, (hipfftComplex *)imCUDA, imCUDA);HANDLE_ERROR_KERNEL;

	//copy result to host
	HANDLE_ERROR(hipMemcpy(convResult,imCUDA,sizeof(imageType)*imSize,hipMemcpyDeviceToHost));

	//release memory
	( hipfftDestroy(fftPlanInv) );HANDLE_ERROR_KERNEL;
    ( hipfftDestroy(fftPlanFwd) );HANDLE_ERROR_KERNEL;
	HANDLE_ERROR( hipFree( imCUDA));
	HANDLE_ERROR( hipFree( kernelCUDA));
	HANDLE_ERROR( hipFree( kernelPaddedCUDA));

	return convResult;
}

//=====================================================================
//WARNING: this library assumes that image and kernel are stored in row major order (C style)
//         BUT the shape of the dimensions are expected to be given as {x-shape,y-shape,z-shape} (not C style)
//WARNING: for cuFFT the fastest running index is z direction!!! so pos = z + imDim[2] * (y + imDim[1] * x)
//NOTE: to avoid transferring a large padded kernel, since memcpy is a limiting factor

 void convolution3DfftCUDAInPlace(imageType* im,int* imDim,imageType* kernel,int* kernelDim,int devCUDA)
{
	hipfftComplex* imCUDA = NULL;
	imageType* kernelCUDA = NULL;
	hipfftComplex* kernelPaddedCUDA = NULL;
	imageType* shifted_kernel = NULL;

	hipfftHandle fftPlanFwd, fftPlanInv;

	
	HANDLE_ERROR( hipSetDevice( devCUDA ) );


	//size of the R2C transform in units of cuFFTComplex
	//should be n_z*n_y*(|_n_x/2_| + 1)*2 = 2*(n_z*n_y*n_x + n_z*n_y)
	std::vector<size_t> math_kernel_shape(kernelDim,kernelDim+3);
	std::vector<size_t> kernel_shape(math_kernel_shape.rbegin(), math_kernel_shape.rend());
	// std::vector<size_t> kernel_shape(kernelDim,kernelDim+3);
	std::vector<size_t> math_stack_shape(imDim,imDim+3);
	std::vector<size_t> stack_shape(math_stack_shape.rbegin(), math_stack_shape.rend());
	//	std::vector<size_t> stack_shape(imDim,imDim+3);
	std::vector<size_t> complex_shape(stack_shape);
	complex_shape[fc::row_major::x] = (stack_shape[fc::row_major::x]/2) + 1;

	const size_t size_krn = std::accumulate(kernel_shape.begin(),
						kernel_shape.end(),
						1,
						std::multiplies<size_t>());
	
	const size_t size_img = std::accumulate(stack_shape.begin(),
						stack_shape.end(),
						1,
						std::multiplies<size_t>());
	
	const size_t size_fft_as_complex = std::accumulate(complex_shape.begin(),
							   complex_shape.end(),
							   1,
							   std::multiplies<long long int>()); 

	const long long int size_fft_as_byte = size_fft_as_complex*sizeof(hipfftComplex); 
	const long long int size_img_as_byte = size_img*sizeof(imageType); 
	const long long int size_krn_as_byte = size_krn*sizeof(imageType); 
	
	HANDLE_ERROR( hipMalloc( (void**)&(shifted_kernel), size_img_as_byte ) );
	HANDLE_ERROR( hipMemset( shifted_kernel, 0, size_img_as_byte ));
	
	HANDLE_ERROR( hipMalloc( (void**)&(kernelCUDA), size_krn_as_byte ) );
	HANDLE_ERROR( hipMemcpy( kernelCUDA, kernel, size_krn_as_byte , hipMemcpyHostToDevice ) );

	int numThreads=std::min((size_t)MAX_THREADS_CUDA,
				size_krn);
	
	numThreads = closest_multiplier(numThreads);
	int numBlocks=std::min((long long int)MAX_BLOCKS_CUDA,
			       (long long int)(size_krn+numThreads-1)/(numThreads));
	fftShiftKernel<<<numBlocks,numThreads>>>(kernelCUDA,
						 shifted_kernel,
						 kernelDim[0],kernelDim[1],kernelDim[2],
						 // imDim[0],imDim[1],imDim[2]
						 stack_shape[fc::row_major::x],
						 stack_shape[fc::row_major::y],
						 stack_shape[fc::row_major::z]
						 );
	HANDLE_ERROR_KERNEL;
	HANDLE_ERROR( hipFree( kernelCUDA));kernelCUDA=NULL;

	HANDLE_ERROR(hipDeviceSynchronize());

	//allocate kernel memory in GPU
	HANDLE_ERROR( hipMalloc( (void**)&(kernelPaddedCUDA), size_fft_as_byte ) );
	HANDLE_ERROR( hipMemset( kernelPaddedCUDA, 0, size_fft_as_byte ));

	float* d_src = 0;
	hipfftComplex* d_dst = 0;
	
	for(size_t z = 0;z<stack_shape[fc::row_major::in_z];++z)
	  for(size_t y = 0;y<stack_shape[fc::row_major::in_y];++y){
	    size_t dst_line_offset = (z*complex_shape[fc::row_major::in_y]*complex_shape[fc::row_major::in_x])+ (y*complex_shape[fc::row_major::in_x]);
	    d_dst = kernelPaddedCUDA+dst_line_offset;
	
	    size_t src_line_offset = (z*stack_shape[fc::row_major::in_y]*stack_shape[fc::row_major::in_x])+ (y*stack_shape[fc::row_major::in_x]);
	    d_src = shifted_kernel + src_line_offset;
	
	    HANDLE_ERROR( hipMemcpy( d_dst ,
				      d_src,
				      stack_shape[fc::row_major::in_x]*sizeof(imageType),
				      hipMemcpyDeviceToDevice ) );
	  }
	

	//make sure GPU finishes 
	HANDLE_ERROR(hipDeviceSynchronize());
	HANDLE_ERROR( hipFree( shifted_kernel ));shifted_kernel=NULL;

	HANDLE_ERROR( hipMalloc( (void**)&(imCUDA), size_fft_as_byte ) );

	std::vector<hipfftComplex> padded_image(size_fft_as_complex);

	float* src_begin = 0;
	hipfftComplex* dst_begin = 0;
	
	for(size_t z = 0;z<stack_shape[fc::row_major::in_z];++z)
	  for(size_t y = 0;y<stack_shape[fc::row_major::in_y];++y){
	    size_t dst_line_offset = (z*complex_shape[fc::row_major::in_y]*complex_shape[fc::row_major::in_x])+ (y*complex_shape[fc::row_major::in_x]);
	    dst_begin = &padded_image[0]+(dst_line_offset);
	
	    size_t src_line_offset = (z*stack_shape[fc::row_major::in_y]*stack_shape[fc::row_major::in_x])+ (y*stack_shape[fc::row_major::in_x]);
	    src_begin = im + src_line_offset;

	    std::copy(src_begin,src_begin + stack_shape[fc::row_major::in_x],(float*)dst_begin);
	    
	  }

	HANDLE_ERROR( hipMemcpy( imCUDA ,
				  &padded_image[0],
				  size_fft_as_byte,
				  hipMemcpyHostToDevice ) );

	
	// THROW_CUFFT_ERROR(hipfftPlan3d(&fftPlanFwd, imDim[0], imDim[1], imDim[2], HIPFFT_R2C));
	THROW_CUFFT_ERROR(hipfftPlan3d(&fftPlanFwd, stack_shape[fc::row_major::z], stack_shape[fc::row_major::y], stack_shape[fc::row_major::x], HIPFFT_R2C));
	
	//TODO: is this needed only cuda 6 or earlier
	
	THROW_CUFFT_ERROR(hipfftExecR2C(fftPlanFwd, (hipfftReal *)imCUDA, imCUDA));
	//transforming image
	THROW_CUFFT_ERROR(hipfftExecR2C(fftPlanFwd, (hipfftReal *)kernelPaddedCUDA, kernelPaddedCUDA));

	numThreads=std::min((size_t)MAX_THREADS_CUDA,size_fft_as_complex);
	long long int chunking = (size_fft_as_complex-1+numThreads)/(numThreads);
	numBlocks=std::min((long long int)MAX_BLOCKS_CUDA,
			   chunking);
	const float scale = 1.0f/(float)(size_img);
	modulateAndNormalize_kernel<<<numBlocks,numThreads>>>(imCUDA,
							      kernelPaddedCUDA,
							      size_fft_as_complex, //size imCUDA
							      scale // scale
							      );HANDLE_ERROR_KERNEL;//last parameter is the size of the FFT


	( hipfftDestroy(fftPlanFwd) );HANDLE_ERROR_KERNEL;
	HANDLE_ERROR( hipFree( kernelPaddedCUDA));

	//inverse FFT 
	// THROW_CUFFT_ERROR(hipfftPlan3d(&fftPlanInv, imDim[0], imDim[1], imDim[2], HIPFFT_C2R));
	THROW_CUFFT_ERROR(hipfftPlan3d(&fftPlanInv, stack_shape[fc::row_major::z], stack_shape[fc::row_major::y], stack_shape[fc::row_major::x], HIPFFT_C2R));

	//TODO: check if this is needed with CUDA 6.*
	THROW_CUFFT_ERROR(hipfftExecC2R(fftPlanInv, imCUDA, (hipfftReal *)imCUDA));
	

	//copy result to host and overwrite image
	HANDLE_ERROR(hipMemcpy(&padded_image[0],
				imCUDA,
				size_fft_as_byte,
				hipMemcpyDeviceToHost));


	//release memory
	( hipfftDestroy(fftPlanInv) );HANDLE_ERROR_KERNEL;
	HANDLE_ERROR( hipFree( imCUDA));

	float* complex_begin = 0;
	float* real_begin = 0;

	//get the right pixel lines again
	for(size_t z = 0;z<stack_shape[fc::row_major::in_z];++z)
	  for(size_t y = 0;y<stack_shape[fc::row_major::in_y];++y){
	    size_t dst_line_offset = (z*stack_shape[fc::row_major::in_y]*stack_shape[fc::row_major::in_x])+ (y*stack_shape[fc::row_major::in_x]);
	    real_begin = &im[0]+dst_line_offset;
	
	    size_t src_line_offset = (z*complex_shape[fc::row_major::in_y]*complex_shape[fc::row_major::in_x])+ (y*complex_shape[fc::row_major::in_x]);
	    complex_begin = (float*)(&padded_image[0] + (src_line_offset));

	    std::copy(complex_begin,complex_begin + stack_shape[fc::row_major::in_x],real_begin);
	    
	  }
	
		
}


int cuda_version(){
  int value = CUDART_VERSION;
  return value;
}


int gpu_mem_needed_mb(int* shape, int len){
  size_t workSize_bytes = 0;
  size_t* ws = &workSize_bytes;

  if(len==1)
    THROW_CUFFT_ERROR(hipfftEstimate1d(shape[0],HIPFFT_R2C,1,ws));

  if(len==2)
    THROW_CUFFT_ERROR(hipfftEstimate2d(shape[0],shape[1],HIPFFT_R2C,ws));

  if(len==3)
    THROW_CUFFT_ERROR(hipfftEstimate3d(shape[0],shape[1],shape[2],HIPFFT_R2C,ws));

  int value = workSize_bytes/(1 << 20);
  return value;
  
}
